
#include <hip/hip_runtime.h>
//==============================================================================
//
//  @@-COPYRIGHT-START-@@
//
//  Copyright (c) 2017-2018, Qualcomm Innovation Center, Inc. All rights reserved.
//
//  Redistribution and use in source and binary forms, with or without
//  modification, are permitted provided that the following conditions are met:
//
//  1. Redistributions of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//  2. Redistributions in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//  3. Neither the name of the copyright holder nor the names of its contributors
//     may be used to endorse or promote products derived from this software
//     without specific prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
//  AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
//  IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
//  ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
//  LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
//  CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
//  SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
//  INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
//  ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
//  POSSIBILITY OF SUCH DAMAGE.
//
//  SPDX-License-Identifier: BSD-3-Clause
//
//  @@-COPYRIGHT-END-@@
//
//==============================================================================

#ifdef GOOGLE_CUDA

#define EIGEN_USE_GPU
#define EIGEN_USE_THREADS

#include "QcQuantizeOpDeprecated.hpp"

using namespace tensorflow;

#define EIGEN_USE_GPU
typedef Eigen::GpuDevice GPUDevice;

// GPU specialization of actual computations.
template <typename T>
struct QcQuantizeDeprecatedFunctor<GPUDevice, T>
{
    /*Operator for const input tensors */
    void operator()(const GPUDevice& d, QcOp::OP_CONFIG_TYPE config, const std::vector<const T*>& in_tensors,
                    const std::vector<size_t>& in_tensor_counts, std::vector<T*> out_tensors,
                    DlQuantization::TfEncodingLayer& in_encoding, DlQuantization::TfEncodingLayer& out_encoding,
                    T* output_min_tensor, T* output_max_tensor, QcOp::QC_Quantizer<T>& quantizer)
    {
        quantizer.Forward(config, in_tensors, in_tensor_counts, out_tensors, in_encoding, out_encoding);

        // copy input_tensors to output_tensors
        // passthrough for CONFIG_TYPE_UPDATE_STATS
        if (config == QcOp::CONFIG_TYPE_UPDATE_STATS)
        {
            for (int idx = 0; idx < in_tensors.size(); idx++)
            {
                cudaMemcpy(out_tensors[idx], in_tensors[idx], in_tensor_counts[idx] * sizeof(T),
                           cudaMemcpyDeviceToDevice);
            }
        }
        long long int enc_size = static_cast<long long int>(out_encoding.out.size());
        T output_min[enc_size], output_max[enc_size];

        // copy min and max out_encodings in local variable separetely
        for (int idx = 0; idx < enc_size; idx++)
        {
            output_min[idx] = out_encoding.out[idx].min;
            output_max[idx] = out_encoding.out[idx].max;
        }
        // transfer min and max encodings to GPU
        cudaMemcpy(output_min_tensor, &output_min, enc_size * sizeof(T), cudaMemcpyHostToDevice);
        cudaMemcpy(output_max_tensor, &output_max, enc_size * sizeof(T), cudaMemcpyHostToDevice);
    }

    /*Operator for non-const input tensors */
    void operator()(const GPUDevice& d, QcOp::OP_CONFIG_TYPE config, std::vector<T*>& in_tensors,
                    const std::vector<size_t>& in_tensor_counts, const bool* training_in_progress,
                    std::vector<T*> out_tensors, DlQuantization::TfEncodingLayer& in_encoding,
                    DlQuantization::TfEncodingLayer& out_encoding, T* output_min_tensor, T* output_max_tensor,
                    QcOp::QC_Quantizer<T>& quantizer)
    {
        // Read the GPU memory to parse the training_in_progress flag
        bool is_train;
        cudaMemcpy(&is_train, training_in_progress, sizeof(is_train), cudaMemcpyDeviceToHost);

        quantizer.Forward(config, in_tensors, in_tensor_counts, is_train, out_tensors, in_encoding, out_encoding);

        // copy input_tensors to output_tensors
        // passthrough for CONFIG_TYPE_UPDATE_STATS
        if (config == QcOp::CONFIG_TYPE_UPDATE_STATS)
        {
            for (int idx = 0; idx < in_tensors.size(); idx++)
            {
                cudaMemcpy(out_tensors[idx], in_tensors[idx], in_tensor_counts[idx] * sizeof(T),
                           cudaMemcpyDeviceToDevice);
            }
        }
        long long int enc_size = static_cast<long long int>(out_encoding.out.size());
        T output_min[enc_size], output_max[enc_size];

        // copy min and max out_encodings in local variable separetely
        for (int idx = 0; idx < enc_size; idx++)
        {
            output_min[idx] = out_encoding.out[idx].min;
            output_max[idx] = out_encoding.out[idx].max;
        }
        // transfer min and max encodings to GPU
        cudaMemcpy(output_min_tensor, &output_min, enc_size * sizeof(T), cudaMemcpyHostToDevice);
        cudaMemcpy(output_max_tensor, &output_max, enc_size * sizeof(T), cudaMemcpyHostToDevice);
    }
};

// Instantiate functors for the types of OpKernels registered.
template struct QcQuantizeDeprecatedFunctor<GPUDevice, float>;

#endif   // GOOGLE_CUDA
